#include <iostream>
#include <vector>
#include <hipDNN.h>
#include "timer.h"
const int H = 1024;
const int W = 1024;
const int C = 3;
const int FH = 3;
const int FW = 3;
const int K = 64;
const int P = 1;

double calculateChecksum(double* O) {
    double checksum = 0.0;
    for (int k = 0; k < K; ++k) {
        for (int x = 0; x < W; ++x) {
            for (int y = 0; y < H; ++y) {
                checksum +=1;// O[k * W * H + x * H + y];
            }
        }
    }
    return checksum;
}
void initImage(double* M) {
  for(int c = 0; c < C; ++c) {
    for (int i = 0; i < W + P; ++i) {
      for (int j = 0; j < H + P; ++j) {
        M[c * W * H + i * H + j] = (double)(c * (i + j));
      }
    }
  }
  for(int c = 0; c < C; c++) {
    for (int i = 0; i < W + P; ++i) {
      int j = H + P;
      M[c * W * H + i * H + j] = (double)(0);
    }
    for (int j = 0; j < W + P; ++j) {
      int i = H + P;
      M[c * W * H + i * H + j] = (double)(0);
    }
  }
}
void initFilter(double* M) {
  for(int k = 0; k < K; ++k) {
    for(int c = 0; c < C; ++c) {
      for (int i = 0; i < FW; ++i) {
        for (int j = 0; j < FH; ++j) {
          M[k * C * FW * FH + c * FW * FH + i * FH + j] = (double)((c + k) * (i + j));
        }
      }
    }
  }
}

int main() {
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // Define tensor sizes and filter dimensions
    int batch_size = 1;

    double* host_output = (double*)malloc(K * W * H * sizeof(double));
    double* host_input = (double*)malloc(C * (W + 2 * P) * (H + 2 * P) * sizeof(double));
    double* host_filter = (double*)malloc(C * K * (FW * FH) * sizeof(double));
    initImage(host_input);
    initFilter(host_filter);
    // Create and set descriptors
    hipdnnTensorDescriptor_t input_descriptor, output_descriptor;
    hipdnnFilterDescriptor_t filter_descriptor;
    hipdnnConvolutionDescriptor_t convolution_descriptor;

    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnCreateFilterDescriptor(&filter_descriptor);
    hipdnnCreateConvolutionDescriptor(&convolution_descriptor);

    hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, batch_size, C, W + 2 * P, H + 2 * P);
    hipdnnSetTensor4dDescriptor(output_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_DOUBLE, batch_size, K, H, W);
    hipdnnSetFilter4dDescriptor(filter_descriptor, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NCHW, K, C, FH, FW);
    hipdnnSetConvolution2dDescriptor(convolution_descriptor,P, P, 1, 1, 2, 2, HIPDNN_CONVOLUTION, HIPDNN_DATA_DOUBLE);

    double *input, *output, *filter;

    hipMalloc((void**)&filter, sizeof(double) * K * C * FH * FW);
    hipMalloc((void**)&input, sizeof(double) * C * (W + 2 * P) * (H + 2 * P));
    hipMalloc((void**)&output, sizeof(double) * K * W * H);

    hipMemcpy(filter, host_filter, sizeof(double) *  K * C * FH * FW, hipMemcpyHostToDevice);
    hipMemcpy(input, host_input, sizeof(double) * C * (W + 2 * P) * (H + 2 * P), hipMemcpyHostToDevice);

    // hipMemcpy(device_filter, filter, sizeof(double) * K * C * FH * FW, hipMemcpyHostToDevice);
    // hipMemcpy(device_input, input, sizeof(double) * C * (W + 2 * P) * (H + 2 * P), hipMemcpyHostToDevice);
    hipdnnConvolutionFwdAlgoPerf_t convolution_algorithm;

    int returnedAlgoCount;
    cudnnGetConvolutionForwardAlgorithm_v7(cudnn, input_descriptor, filter_descriptor, convolution_descriptor, \
                                        output_descriptor, 1, &returnedAlgoCount, &convolution_algorithm);
    hipdnnConvolutionFwdAlgo_t algo = convolution_algorithm.algo;

    double alpha = 1.0f, beta = 0.0f;

    hipEvent_t start, stop;
    initialize_timer();
    start_timer();

    hipdnnConvolutionForward(cudnn, &alpha, input_descriptor, input, filter_descriptor, filter, \
    convolution_descriptor, algo, nullptr, 0, &beta, output_descriptor, output);
    stop_timer();

    hipMemcpy(host_output, output, sizeof(double) * K * W * H, hipMemcpyDeviceToHost);

    double time = elapsed_time();
    double checkSum = calculateChecksum(host_output);
    // double checkSum = 0;
    printf( "checkSum: %lf\ntime: %lf\n", checkSum,  time*1000);
    
    hipError_t cudaStatus = hipGetLastError();
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaStatus));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaStatus));
        // Handle error accordingly
    }



    // Cleanup
    hipFree(input);
    hipFree(output);
    hipFree(filter);
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(filter_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
    hipdnnDestroy(cudnn);

    return 0;
}
