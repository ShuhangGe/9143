#include <iostream>
#include <vector>
#include <hipDNN.h>
#include <hip/hip_runtime.h>

float CheckSum(double* M) {
  float result = 0;
  for(int k = 0; k < K; k++) {
    for (int i = 0; i < W; ++i) {
      for (int j = 0; j < H; ++j) {
        result += M[k * W * H + i * H + j];
      }
    }
  }
  return result;
}

int main() {
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    // Define tensor sizes and filter dimensions
    // Example: input = 1x3x128x128, filter = 10x3x3x3
    int batch_size = 1, channels = 3, height = 1024, width = 1024;
    int filter_height = 3, filter_width = 3, output_channels = 64;

    // Create and set descriptors
    hipdnnTensorDescriptor_t input_descriptor, output_descriptor;
    hipdnnFilterDescriptor_t filter_descriptor;
    hipdnnConvolutionDescriptor_t convolution_descriptor;

    hipdnnCreateTensorDescriptor(&input_descriptor);
    hipdnnCreateTensorDescriptor(&output_descriptor);
    hipdnnCreateFilterDescriptor(&filter_descriptor);
    hipdnnCreateConvolutionDescriptor(&convolution_descriptor);

    hipdnnSetTensor4dDescriptor(input_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, batch_size, channels, height, width);
    hipdnnSetFilter4dDescriptor(filter_descriptor, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, output_channels, channels, filter_height, filter_width);
    hipdnnSetConvolution2dDescriptor(convolution_descriptor, 0, 0, 1, 1, 1, 1, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT);

    // Find the dimensions of the output tensor
    int n, c, h, w;
    hipdnnGetConvolution2dForwardOutputDim(convolution_descriptor, input_descriptor, filter_descriptor, &n, &c, &h, &w);
    hipdnnSetTensor4dDescriptor(output_descriptor, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, n, c, h, w);

    // Allocate memory for input, output, and filter
    float *input, *output, *filter;
    hipMalloc(&input, batch_size * channels * height * width * sizeof(float));
    hipMalloc(&output, n * c * h * w * sizeof(float));
    hipMalloc(&filter, output_channels * channels * filter_height * filter_width * sizeof(float));

    // Initialize memory for input and filter (omitted for brevity)

    // Selecting the convolution algorithm (HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST)
    hipdnnConvolutionFwdAlgoPerf_t convolution_algorithm;
    hipdnnConvolutionFwdAlgo_t algo = convolution_algorithm.algo;

    int returnedAlgoCount;
    cudnnGetConvolutionForwardAlgorithm_v7(cudnn, input_descriptor, filter_descriptor, convolution_descriptor, output_descriptor, 1, &returnedAlgoCount, &convolution_algorithm);

    // Allocate workspace for the selected algorithm
    size_t workspace_bytes = 0;
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn, input_descriptor, filter_descriptor, convolution_descriptor, output_descriptor, algo, &workspace_bytes);
    void *workspace = nullptr;
    if (workspace_bytes > 0) {
        hipMalloc(&workspace, workspace_bytes);
    }

    // Perform the convolution
    float alpha = 1.0f, beta = 0.0f;

    initialize_timer();
    start_timer();
    hipdnnConvolutionForward(cudnn, &alpha, input_descriptor, input, filter_descriptor, filter, convolution_descriptor, algo, workspace, workspace_bytes, &beta, output_descriptor, output);
    stop_timer();

    double time = elapsed_time();
    float checkSum = CheckSum(output);
    // float checkSum = 0;
    printf( "%lf, %lf, %lf\n", checkSum, time*1000);
    
    hipError_t cudaStatus = hipGetLastError();
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaStatus));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaStatus));
        // Handle error accordingly
    }
    // Print the kernel execution time
    std::cout << "Kernel Execution Time: " << milliseconds << " ms" << std::endl;

    // Compute checksum (sum of elements in 'output')
    // ...

    // Cleanup
    if (workspace) hipFree(workspace);
    hipFree(input);
    hipFree(output);
    hipFree(filter);
    hipdnnDestroyTensorDescriptor(input_descriptor);
    hipdnnDestroyTensorDescriptor(output_descriptor);
    hipdnnDestroyFilterDescriptor(filter_descriptor);
    hipdnnDestroyConvolutionDescriptor(convolution_descriptor);
    hipdnnDestroy(cudnn);

    return 0;
}
